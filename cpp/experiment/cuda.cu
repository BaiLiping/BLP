

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define SIZE 1024*1024*1024
__global__ void VectorAdd(int *a, int *b, int *c, int n){
//void VectorAdd(int *a, int *b, int *c, int n){ 
  int i=threadIdx.x;
    if ( i<n-10){
        a[i]=b[i]*a[i+1]*a[i+2];
        b[i]=a[i+1]+b[i+1]*a[i+4];
        c[i]=a[i+1]*a[i]*a[i+1]*b[i]*b[i+1];
    }
    if(i<n-10){
        c[i]=c[i+1]*c[i+2]*c[i+3]*c[i+4];
    }

}

int main(){
    int *a, *b, *c;
    int *d_a,*d_b,*d_c;
a=(int *) malloc (SIZE*sizeof(int));    
b=(int *) malloc (SIZE*sizeof(int));
c=(int *) malloc (SIZE*sizeof(int));

hipMalloc(&d_a,SIZE*sizeof(int));
hipMalloc(&d_b,SIZE*sizeof(int));
hipMalloc(&d_c,SIZE*sizeof(int));

for (int i=0;i<SIZE;++i){
    a[i]*=i;
    b[i]*=i;
    c[i]=0;
}
hipMemcpy(d_a, a, SIZE*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_b,b,SIZE*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(d_c,c, SIZE*sizeof(int),hipMemcpyHostToDevice);

VectorAdd<<<1,SIZE>>>(d_a,d_b,d_c,SIZE);
//VectorAdd(a,b,c,SIZE);

hipMemcpy(c,d_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);
//for (int i=0;i<100;++i)
//printf("c[%d]=%d\n",i,c[i]);
free(a);
free(b);
free(c);
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
